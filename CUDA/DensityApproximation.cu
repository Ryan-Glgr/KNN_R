#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>

//device function to partition array
__device__ int partition(double* dist, int left, int right) {
    double pivot = dist[right];  //use rightmost element as pivot
    int i = left - 1;  //pointer to the smaller element

    //swap elements around pivot
    for (int j = left; j <= right - 1; j++) {
        if (dist[j] <= pivot) {
            i++;
            // Swap dist[i] and dist[j]
            double temp = dist[i];
            dist[i] = dist[j];
            dist[j] = temp;
        }
    }

    //swap dist[i + 1] and dist[right]
    double temp = dist[i + 1];
    dist[i + 1] = dist[right];
    dist[right] = temp;

    return (i + 1);  // Return partition index
}

//device function to get distance to k-th nearest element
__device__ double quickselect(double* dist, int left, int right, int k) {
    if (left == right) {
        return dist[left];
    }

    int pivotIndex = partition(dist, left, right);
    
    if (k == pivotIndex) {
        return dist[k];  //return value of k-th smallest element
    } else if (k < pivotIndex) { //search right side of pivot
        return quickselect(dist, left, pivotIndex - 1, k);
    } else { //search left side of pivot
        return quickselect(dist, pivotIndex + 1, right, k);
    }
}

//computes density of single point for each trhead
__global__ void kNNDensity(const double* data, double* result, int N, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; //get index
    
    if (i < N) {
        //get distances
        double dist[N]; //allocate local memory to store distances
        for(int j = 0; j < N; j++)
            dist[j] = fabs(data[i] - data[j]);

        //get distance to k-th nearest neighbor
        double Ri = quickselect(dist, 0, N - 1, k);

        //compute density approximation
        result[i] = static_cast<double>(k) / (N * 2 * Ri);
    }
}

//wrapper function to handle CUDA execution
std::vector<double> getDensity(const std::vector<double>& data, int k) {
    int N = data.size();
    
    //set the value of k to N-1 in the case where k > N-1
    if(k > N - 1)
        k = N - 1;

    //allocate device memory
    double *d_data, /* *d_dist, */ *d_result;
    hipMalloc(&d_data, N * sizeof(double)); //store input data
    hipMalloc(&d_result, N * sizeof(double)); //store the result for each data point
    
    //copy input data to device
    hipMemcpy(d_data, data.data(), N * sizeof(double), hipMemcpyHostToDevice);

    //launch CUDA kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    
    //calculate each elements density approximation in individual threads
    kNNDensity<<<numBlocks, blockSize>>>(d_data, d_result, N, k);
    hipDeviceSynchronize(); //wait for all threads to finish execution

    //copy result back to host
    std::vector<double> host_result(N);
    hipMemcpy(host_result.data(), d_result, N * sizeof(double), hipMemcpyDeviceToHost);

    //free device memory
    hipFree(d_data);
    hipFree(d_result);

    return host_result;
}

void printVector(const std::vector<double>& vec) {
    std::cout << "[ ";
    for (const double& val : vec) {
        std::cout << val << " ";
    }
    std::cout << "]" << std::endl;
}

int main() {
    std::vector<double> data = {3.0, 3.0, 20.0, 17.0, 7.0};
    printf("input:\n");
    printVector(data);

    std::vector<double> density = getDensity(data, 2);
    printf("density result:\n");
    printVector(density);
}